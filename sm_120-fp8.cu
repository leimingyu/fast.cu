#include "hip/hip_runtime.h"
//----------------------------------------------------------------------------//
// 5th gen tensor core for fp8 on RTX 5080 (blackwell, sm120)
//
// ptx :    tcgen05.mma.cta_group::1.kind::f8f6f4
// tile:    M64N8K32
//
// more details : https://confluence.qualcomm.com/confluence/display/TENSOR/Microscaling+for+FP8
//----------------------------------------------------------------------------//


#include <assert.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <vector>
#include <cstdlib>
#include <iomanip> // setfill, setw
#include <cstdint> // int8_t, uint8_t


#include <hip/hip_runtime.h>
#include <mma.h>
#include <cudaTypedefs.h>
#include <cuda/barrier>
#include <hip/hip_runtime.h>
#include <cuda/pipeline>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h> // FP8


#define DEBUG 1


#define K32 32


enum class FP8Format
{
    E5M2,
    E4M3
};


//----------------------------------------------------------------------------//
// Utility
//----------------------------------------------------------------------------//
void logMessage(const char *format, ...)
{
    // Get the current timestamp
    std::time_t now = std::time(nullptr);
    char timestamp[20];
    std::strftime(timestamp, sizeof(timestamp), "%Y-%m-%d %H:%M:%S", std::localtime(&now));


    // Print the timestamp
    std::cout << "[" << timestamp << "] ";


    // Handle the variadic arguments
    va_list args;
    va_start(args, format);
    vprintf(format, args); // Print the formatted message
    va_end(args);


    // End with a newline
    std::cout << std::endl;
}


void cudaCheck(hipError_t error, const char *file, int line)
{
    if (error != hipSuccess)
    {
        printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
               hipGetErrorString(error));
        exit(1);
    }
}


#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))


//----------------------------------------------------------------------------//
// Functions
//----------------------------------------------------------------------------//
void printProgressBar(double percentage);


template <int TILE_K, FP8Format FORMAT>
void runTest(std::vector<std::vector<uint8_t>> &current_test_ab,
             std::vector<uint32_t> &current_test_c,
             std::vector<uint32_t> &current_result,
             int SM_num);


//----------------------------------------------------------------------------//
// gpu kernel
//----------------------------------------------------------------------------//


//----------------------------------------------------------------------------//
//
//----------------------------------------------------------------------------//
int main(int argc, char **argv)
{
    //------------------------------------------------------------------------//
    // Read commandlines
    //------------------------------------------------------------------------//
    if (argc != 4)
    {
        std::cerr << "\nUsage: " << argv[0] << " <filename> <format> <device_id>" << std::endl;
        std::cerr << "  format: e5m2 or e4m3" << std::endl;
        std::cerr << "  device_id: GPU device ID (0 to N-1)" << std::endl;
        return 1;
    }


    // Parse the format argument
    std::string format_str = argv[2];
    bool use_e5m2;
    if (format_str == "e5m2")
    {
        use_e5m2 = true;
    }
    else if (format_str == "e4m3")
    {
        use_e5m2 = false;
    }
    else
    {
        std::cerr << "Error: format must be either 'e5m2' or 'e4m3'" << std::endl;
        return 1;
    }


    // Parse and set the device ID
    int device_id = std::stoi(argv[3]);
    int device_count;
    cudaCheck(hipGetDeviceCount(&device_count));


    if (device_id < 0 || device_id >= device_count)
    {
        std::cerr << "Error: device_id must be between 0 and " << (device_count - 1) << std::endl;
        return 1;
    }


    cudaCheck(hipSetDevice(device_id));


    // Get and print device properties
    hipDeviceProp_t prop;
    cudaCheck(hipGetDeviceProperties(&prop, device_id));
    logMessage("Using GPU device %d: %s", device_id, prop.name);
    logMessage("Number of SMs: %d\n", prop.multiProcessorCount);


    int SM_NUM = prop.multiProcessorCount;


    //------------------------------------------------------------------------//
    // Read all test cases
    //------------------------------------------------------------------------//
    // 65 input values per row:   c +  32 of a/b for K32 case
    // 33 input values per row:   c +  16 of a/b for K16 case
    // 17 input values per row:   c +   8 of a/b for  K8 case
    std::vector<uint32_t> allTests_c;              // input c in fp32
    std::vector<std::vector<uint8_t>> allTests_ab; // input a/b in fp8


    std::cout << "file : " << argv[1] << std::endl;


    std::ifstream file(argv[1], std::ifstream::ate | std::ifstream::binary);
    if (!file.is_open())
    {
        std::cerr << "Unable to open file " << argv[1] << std::endl;
        return 1;
    }


    // Get the size of the file
    std::streamsize fileSize = file.tellg();
    file.seekg(0, std::ios::beg);


    std::cout << "Read all the tests cases ... " << std::endl;


    std::string line;
    std::streamsize totalRead = 0;
    size_t lineCount = 0;


    while (getline(file, line))
    {
        // print progress
        lineCount++;
        totalRead += line.size() + 1;
        if (lineCount % 100 == 0)
            printProgressBar(static_cast<double>(totalRead) / fileSize);


        // read line :    c a0 b0 a1 b1 ....  aN bN
        std::istringstream iss(line);
        std::vector<uint8_t> numbers_ab; // current line for a/b
        std::string hexStr;


        // read c first
        iss >> hexStr;
        uint32_t num_c = static_cast<uint32_t>(std::stoul(hexStr, nullptr, 16)); // was uint16_t
        allTests_c.push_back(num_c);                                             // store current line for C


        // read a/b
        while (iss >> hexStr)
        {
            uint8_t num = static_cast<uint8_t>(std::stoul(hexStr, nullptr, 16));
            numbers_ab.push_back(num);
        }


        // Check if this is a K8/K16/K32 case
        if (numbers_ab.size() == 32) // K16
        {
            numbers_ab.resize(64, 0); // Add 32 zeros
        }
        else if (numbers_ab.size() == 16)
        {
            numbers_ab.resize(64, 0); // Add zeros
        }
        else if (numbers_ab.size() != 64)
        {
            std::cerr << "Error: Invalid input line length. Expected 16 or 32 or 64 values for A/B, got "
                      << numbers_ab.size() << std::endl;
            return 1;
        }


        allTests_ab.push_back(numbers_ab); // store current line for a and b
    }


    std::cout << std::endl;


    file.close();


    //------------------------------------------------------------------------//
    // Check first line : c + 32x{a(i), b(i)}
    //------------------------------------------------------------------------//
    printf("\nCheck first line of input file:\n");
    printf("%08X ", allTests_c[0]);
    for (int i = 0; i < 64; i++)
    {
        printf("%02X ", allTests_ab[0][i]);
    }
    printf("\n");


    //------------------------------------------------------------------------//
    // Run all test cases
    //------------------------------------------------------------------------//
    std::cout << "Run Tensor Core Tests with " << format_str << " format\n"
              << std::endl;


    // Change from int to size_t for totalNum
    size_t totalNum = allTests_ab.size();


    // results in fp32
    std::vector<uint32_t> allTests_results(totalNum);


    size_t batches = (totalNum + SM_NUM - 1) / SM_NUM;


    for (size_t i = 0; i < batches; ++i)
    {
        size_t start_idx = i * SM_NUM;
        size_t end_idx = std::min(start_idx + SM_NUM, totalNum);


        if ((end_idx % 1000) == 0)
            logMessage("case : %zu (%zu : %.2f %% done) \n", end_idx, totalNum, (end_idx / (float)totalNum) * 100);


        size_t test_counts = end_idx - start_idx;


        // Create tests for this batch from allTests_c and allTests_ab.
        std::vector<uint32_t> partTests_c(allTests_c.begin() + start_idx, allTests_c.begin() + end_idx);
        std::vector<std::vector<uint8_t>> partTests_ab(allTests_ab.begin() + start_idx, allTests_ab.begin() + end_idx);


        // output in fp32
        std::vector<uint32_t> current_result(test_counts);


        //--------------------------------------------------------------------//
        // run tensor core test
        //--------------------------------------------------------------------//
        if (use_e5m2)
        { // Add a command line argument or configuration to set this
            runTest<K32, FP8Format::E5M2>(partTests_ab, partTests_c, current_result, SM_NUM);
        }
        else
        {
            // runTest<K32, FP8Format::E4M3>(partTests_ab, partTests_c, current_result);
        }


        // update the test results
        std::copy(current_result.begin(), current_result.end(), allTests_results.begin() + start_idx);
    }


    //------------------------------------------------------------------------//
    // Export the results
    //------------------------------------------------------------------------//
    // std::string outFileName = "gpu_output.txt";
    // std::ofstream outFile(outFileName);
    // if (!outFile)
    // {
    //     std::cerr << "Error opening file to write." << std::endl;
    //     return 1;
    // }


    // const int colNum = 2;
    // for (int i = 0; i < totalNum; i++)
    // {
    //     for (int j = 0; j < colNum; j++)
    //     {
    //         // outFile << allTests_results[i][j];
    //         outFile << std::setfill('0') << std::setw(8) << std::hex << allTests_results[i][j];
    //         if (j < colNum)
    //         {
    //             outFile << " "; // separate with space
    //         }
    //     }
    //     outFile << "\n"; // EOR
    // }


    // outFile.close();


    // std::cout << "\nResults are saved! Check " << outFileName << ".\n";


    return 0;
}


//----------------------------------------------------------------------------//
// print read input progress
//----------------------------------------------------------------------------//
void printProgressBar(double percentage)
{
    int barWidth = 50;
    std::cout << "[";
    int pos = static_cast<int>(barWidth * percentage);
    for (int i = 0; i < barWidth; ++i)
    {
        if (i < pos)
            std::cout << "=";
        else if (i == pos)
            std::cout << ">";
        else
            std::cout << " ";
    }
    std::cout << "] " << int(percentage * 100.0) << " %\r";
    std::cout.flush();
}


//----------------------------------------------------------------------------//
// host code to prepare test
//----------------------------------------------------------------------------//
template <int TILE_K, FP8Format FORMAT>
void runTest(std::vector<std::vector<uint8_t>> &current_test_ab,
             std::vector<uint32_t> &current_test_c,
             std::vector<uint32_t> &current_result,
             int SM_num)
{
    int test_batch_size = current_test_ab.size();

    std::cout << "test size :  " << test_batch_size << std::endl;
    std::cout << "using SM  =   " << SM_num << std::endl;


    //------------------------------------------------------------------------//
    // mma size :  M 64 x N 8 x K 32
    //------------------------------------------------------------------------//
    long M = 64, N = 8, K = 32;


    //------------------------------------------------------------------------//
    // prepare host buffers
    //------------------------------------------------------------------------//
    uint8_t *hA = nullptr;
    uint8_t *hB = nullptr;

    uint32_t *hD = nullptr;
    uint32_t *hresult = nullptr;

    size_t sizeA = M * K * SM_num;
    size_t sizeB = N * K * SM_num;
    size_t sizeD = M * N * SM_num;   // noted: for fp16 accumulation, each 32 reg will store two fp16 results

    hA = (uint8_t *)malloc(sizeof(uint8_t) * sizeA);
    hB = (uint8_t *)malloc(sizeof(uint8_t) * sizeB);
    hD = (uint32_t *)malloc(sizeof(uint32_t) * sizeD);
    hresult = (uint32_t *)malloc(sizeof(uint32_t) * sizeD);

    // init to 0
    memset(hA, 0, sizeof(uint8_t) * sizeA);     // MxKxSM_num
    memset(hB, 0, sizeof(uint8_t) * sizeB);     // NxKxSM_num
    memset(hD, 0, sizeof(uint32_t) * sizeD);    // MxNxSM_num
    memset(hresult, 0, sizeof(uint32_t) * sizeD);


    //------------------------------------------------------------------------//
    // read/set up data on cpu
    //------------------------------------------------------------------------//
    // read a/b for each test case in the batch
    for (int test = 0; test < test_batch_size; test++) {

        // Calculate base offsets for this test case
        size_t a_offset = test * M * K;  // Offset in hA array
        size_t b_offset = test * N * K;  // Offset in hB array
        size_t d_offset = test * M * N;  // Offset in hD array

        // Read a/b pairs for this test case
        for (int i = 0; i < TILE_K; i++) {
            // MxK: read 'a' values
            hA[a_offset + i] = current_test_ab[test][i * 2];
            // NxK: read 'b' values
            hB[b_offset + i] = current_test_ab[test][i * 2 + 1];
        }

        // Read input C for this test case:  1st element of hD, with step size of MxN
        hD[d_offset] = (uint32_t)current_test_c[test];
    }




// #if DEBUG
//     std::cout << "Read input C" << std::endl;
// #endif


// #if DEBUG
//     printf("Pack input C (fp32) : %08X \n\n", hD[0]);
// #endif




    //------------------------------------------------------------------------//
    // gpu buffer
    //------------------------------------------------------------------------//
    uint8_t *dA = nullptr;
    uint8_t *dB = nullptr;
    uint32_t *dD = nullptr;

    hipMalloc((void **)&dA, sizeof(uint8_t) * sizeA);
    hipMalloc((void **)&dB, sizeof(uint8_t) * sizeB);
    hipMalloc((void **)&dD, sizeof(uint32_t) * sizeD);

    // h2d : copy input ops to gpu
    hipMemcpy(dA, hA, sizeof(uint8_t) * sizeA, hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, sizeof(uint8_t) * sizeB, hipMemcpyHostToDevice);
    hipMemcpy(dD, hD, sizeof(uint32_t) * sizeD, hipMemcpyHostToDevice);


    // Launch a single block with 128 threads => "1 warpgroup" (4 warps, 32 threads per warp)
    //matmul_fp8_64x8x32_kernel<FORMAT><<<1, 128>>>(dA, dB, dD);


    // d2h : copy results back to host
    hipMemcpy(hresult, dD, sizeof(uint32_t) * sizeD, hipMemcpyDeviceToHost);

    // note: read the 1st element of each MxN block

    /*
    uint32_t c0 = hresult[0];
    // uint16_t c0_lo = static_cast<uint16_t>(c0 & 0xFFFF);   // read the lower half (1st 16 bits)
    // uint16_t c0_hi = static_cast<uint16_t>((c0 >> 16) & 0xFFFF);


// check value
#if DEBUG
    printf("%08X\n", hresult[0]);
#endif


    //printf("\n\n");


    current_result.push_back(c0);


    hipFree(dA);
    hipFree(dB);
    hipFree(dD);


    free(hA);
    free(hB);
    free(hD);
    free(hresult);


    hipDeviceSynchronize();
    */
}
