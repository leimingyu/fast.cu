#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cudaTypedefs.h>
#include <cuda/barrier>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>


#include <iostream>
#include <fstream>
#include <sstream>

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <unistd.h>
#include <ctime>
#include <vector>
#include <random>
#include <cassert>
#include <unistd.h>

#include <iomanip>  // setfill, setw
#include <cstdint>  // int8_t/uint8_t

#define DEBUG 1
#define K32 32 


#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

void cudaCheck(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
           hipGetErrorString(error));
    exit(1);
  }
}

// typedef __hip_bfloat16 bf16;
#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

template <int TILE_K>
void runTest(std::vector<uint8_t> current_test_ab,
             uint32_t current_test_c,
             std::vector<uint32_t> &current_result);

//----------------------------------------------------------------------------//
// wgmma related apis 
//----------------------------------------------------------------------------//

// wgmma example in bf16
// #include "examples/matmul/matmul_2.cuh"

using barrier = cuda::barrier<cuda::thread_scope_block>;
namespace cde = cuda::device::experimental;

__device__ static inline uint64_t matrix_descriptor_encode(uint64_t x) { return (((x) & 0x3FFFF) >> 0x4); }

// __device__ uint64_t make_smem_desc(bf16* ptr) {
//     uint32_t addr = static_cast<uint32_t>(__cvta_generic_to_shared(ptr));
//     uint64_t desc = 0x0000000000000000;
//     desc |= matrix_descriptor_encode(addr);
//     desc |= matrix_descriptor_encode((uint64_t)16) << 16;
//     desc |= matrix_descriptor_encode((uint64_t)1024) << 32;
//     desc |= 1llu << 62; // 128B swizzle
//     return desc;
//   }

__device__ void warpgroup_arrive() {
    asm volatile("wgmma.fence.sync.aligned;\n" ::: "memory");
}

__device__ void warpgroup_commit_batch() {
    asm volatile("wgmma.commit_group.sync.aligned;\n" ::: "memory");
}

template <int N>
__device__ void warpgroup_wait() {
    static_assert(N >= 0 && N <= 7, "WGMMA wait: N must be in range [0, 7]");
    asm volatile("wgmma.wait_group.sync.aligned %0;\n" ::"n"(N) : "memory");
}

// template <int BlockMajorSize, int BlockMinorSize>
// void create_tensor_map(CUtensorMap *tma_map, bf16* gmem_ptr, int blocks_height, int blocks_width) {
//     void* gmem_address = (void*)gmem_ptr;
//     uint64_t gmem_prob_shape[5] = {(uint64_t)BlockMinorSize*blocks_width, (uint64_t)BlockMajorSize*blocks_height, 1, 1, 1};
//     uint64_t gmem_prob_stride[5] = {sizeof(bf16), sizeof(bf16) * BlockMinorSize*blocks_width, 0, 0, 0};
//     uint32_t smem_box_shape[5] = {uint32_t(BlockMinorSize), uint32_t(BlockMajorSize), 1, 1, 1};
//     uint32_t smem_box_stride[5] = {1, 1, 1, 1, 1};

//     hipError_t result = cuTensorMapEncodeTiled(
//         tma_map, CU_TENSOR_MAP_DATA_TYPE_BFLOAT16, 2, gmem_address, gmem_prob_shape,
//         gmem_prob_stride + 1, smem_box_shape, smem_box_stride, CU_TENSOR_MAP_INTERLEAVE_NONE,
//         CU_TENSOR_MAP_SWIZZLE_128B, CU_TENSOR_MAP_L2_PROMOTION_NONE, CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE);

//     assert(result == hipSuccess);
// }



// template <int BlockMajorSize, int BlockMinorSize>
// __host__ static inline CUtensorMap* allocate_and_create_tensor_map(bf16* src, int blocks_height, int blocks_width) {
//     CUtensorMap *tma_map_d;
//     hipMalloc(&tma_map_d, sizeof(CUtensorMap));
//     CUtensorMap tma_map_host;
//     create_tensor_map<BlockMajorSize, BlockMinorSize>(&tma_map_host, src, blocks_height, blocks_width);
//     hipMemcpy(tma_map_d, &tma_map_host, sizeof(CUtensorMap), hipMemcpyHostToDevice);
//     return tma_map_d;
// }

//----------------------------------------------------------------------------//
// gpu tests and kernels
//----------------------------------------------------------------------------//



/*
__global__ void kernel_wgmma_FP8(float *buf_fp32, half *buf_fp16,
                               uint8_t *test_ops, uint32_t *test_init_c, uint32_t *result_gpu)
{
  //------------------------------------------------------------------------//
  // registers for mma
  //------------------------------------------------------------------------//
  float D[4] = {0.f, 0.f, 0.f, 0.f};
  float C[4] = {0.f, 0.f, 0.f, 0.f};
  uint32_t A[4]; // A0, A1, A2, A3
  uint32_t B[2];

  //------------------------------------------------------------------------//
  // Init A/B to zeros
  //------------------------------------------------------------------------//
  A[0] = 0;
  A[1] = 0;
  A[2] = 0;
  A[3] = 0;

  B[0] = 0;
  B[1] = 0;

  //------------------------------------------------------------------------//
  // first mma call :   C = 0
  //------------------------------------------------------------------------//

  //------------------------------------------------------------------------//
  // Init inputs
  //------------------------------------------------------------------------//
  uint8_t a0, a1, a2, a3, a4, a5, a6, a7, a8, a9, a10, a11, a12, a13, a14, a15, a16, a17, a18, a19, a20, a21, a22, a23, a24, a25, a26, a27, a28, a29, a30, a31;
  uint8_t b0, b1, b2, b3, b4, b5, b6, b7, b8, b9, b10, b11, b12, b13, b14, b15, b16, b17, b18, b19, b20, b21, b22, b23, b24, b25, b26, b27, b28, b29, b30, b31;

  // set a to 0
  a0 = 0;
  a1 = 0;
  a2 = 0;
  a3 = 0;
  a4 = 0;
  a5 = 0;
  a6 = 0;
  a7 = 0;
  a8 = 0;
  a9 = 0;
  a10 = 0;
  a11 = 0;
  a12 = 0;
  a13 = 0;
  a14 = 0;
  a15 = 0;
  a16 = 0;
  a17 = 0;
  a18 = 0;
  a19 = 0;
  a20 = 0;
  a21 = 0;
  a22 = 0;
  a23 = 0;
  a24 = 0;
  a25 = 0;
  a26 = 0;
  a27 = 0;
  a28 = 0;
  a29 = 0;
  a30 = 0;
  a31 = 0;

  // set b to 0
  b0 = 0;
  b1 = 0;
  b2 = 0;
  b3 = 0;
  b4 = 0;
  b5 = 0;
  b6 = 0;
  b7 = 0;
  b8 = 0;
  b9 = 0;
  b10 = 0;
  b11 = 0;
  b12 = 0;
  b13 = 0;
  b14 = 0;
  b15 = 0;
  b16 = 0;
  b17 = 0;
  b18 = 0;
  b19 = 0;
  b20 = 0;
  b21 = 0;
  b22 = 0;
  b23 = 0;
  b24 = 0;
  b25 = 0;
  b26 = 0;
  b27 = 0;
  b28 = 0;
  b29 = 0;
  b30 = 0;
  b31 = 0;

  //--- a0 x b0 ---//
  a0 = (a0 | test_ops[0]);
  b0 = (b0 | test_ops[1]);
  //--- a1 x b1 ---//
  a1 = (a1 | test_ops[2]);
  b1 = (b1 | test_ops[3]);
  //--- a2 x b2 ---//
  a2 = (a2 | test_ops[4]);
  b2 = (b2 | test_ops[5]);
  //--- a3 x b3 ---//
  a3 = (a3 | test_ops[6]);
  b3 = (b3 | test_ops[7]);

  //--- a4 x b4 ---//
  a4 = (a4 | test_ops[8]);
  b4 = (b4 | test_ops[9]);
  //--- a5 x b5 ---//
  a5 = (a5 | test_ops[10]);
  b5 = (b5 | test_ops[11]);
  //--- a6 x b6 ---//
  a6 = (a6 | test_ops[12]);
  b6 = (b6 | test_ops[13]);
  //--- a7 x b7 ---//
  a7 = (a7 | test_ops[14]);
  b7 = (b7 | test_ops[15]);

  //--- a8 x b8 ---//
  a8 = (a8 | test_ops[16]);
  b8 = (b8 | test_ops[17]);
  //--- a9 x b9 ---//
  a9 = (a9 | test_ops[18]);
  b9 = (b9 | test_ops[19]);
  //--- a10 x b10 ---//
  a10 = (a10 | test_ops[20]);
  b10 = (b10 | test_ops[21]);
  //--- a11 x b11 ---//
  a11 = (a11 | test_ops[22]);
  b11 = (b11 | test_ops[23]);

  //--- a12 x b12 ---//
  a12 = (a12 | test_ops[24]);
  b12 = (b12 | test_ops[25]);
  //--- a13 x b13 ---//
  a13 = (a13 | test_ops[26]);
  b13 = (b13 | test_ops[27]);
  //--- a14 x b14 ---//
  a14 = (a14 | test_ops[28]);
  b14 = (b14 | test_ops[29]);
  //--- a15 x b15 ---//
  a15 = (a15 | test_ops[30]);
  b15 = (b15 | test_ops[31]);

  //--- a16 x b16 ---//
  a16 = (a16 | test_ops[32]);
  b16 = (b16 | test_ops[33]);
  //--- a17 x b17 ---//
  a17 = (a17 | test_ops[34]);
  b17 = (b17 | test_ops[35]);
  //--- a18 x b18 ---//
  a18 = (a18 | test_ops[36]);
  b18 = (b18 | test_ops[37]);
  //--- a19 x b19 ---//
  a19 = (a19 | test_ops[38]);
  b19 = (b19 | test_ops[39]);

  //--- a20 x b20 ---//
  a20 = (a20 | test_ops[40]);
  b20 = (b20 | test_ops[41]);
  //--- a21 x b21 ---//
  a21 = (a21 | test_ops[42]);
  b21 = (b21 | test_ops[43]);
  //--- a22 x b22 ---//
  a22 = (a22 | test_ops[44]);
  b22 = (b22 | test_ops[45]);
  //--- a23 x b23 ---//
  a23 = (a23 | test_ops[46]);
  b23 = (b23 | test_ops[47]);

  //--- a24 x b24 ---//
  a24 = (a24 | test_ops[48]);
  b24 = (b24 | test_ops[49]);
  //--- a25 x b25 ---//
  a25 = (a25 | test_ops[50]);
  b25 = (b25 | test_ops[51]);
  //--- a26 x b26 ---//
  a26 = (a26 | test_ops[52]);
  b26 = (b26 | test_ops[53]);
  //--- a27 x b27 ---//
  a27 = (a27 | test_ops[54]);
  b27 = (b27 | test_ops[55]);

  //--- a28 x b28 ---//
  a28 = (a28 | test_ops[56]);
  b28 = (b28 | test_ops[57]);
  //--- a29 x b29 ---//
  a29 = (a29 | test_ops[58]);
  b29 = (b29 | test_ops[59]);
  //--- a30 x b30 ---//
  a30 = (a30 | test_ops[60]);
  b30 = (b30 | test_ops[61]);
  //--- a31 x b31 ---//
  a31 = (a31 | test_ops[62]);
  b31 = (b31 | test_ops[63]);

  // Pass the input ops to tc inputs
  uint8_t a0_t0[4] = {a0, a1, a2, a3};     // A[0] for T0
  uint8_t a0_t1[4] = {a4, a5, a6, a7};     // A[0] for T1
  uint8_t a0_t2[4] = {a8, a9, a10, a11};   // A[0] for T2
  uint8_t a0_t3[4] = {a12, a13, a14, a15}; // A[0] for T3

  uint8_t a2_t0[4] = {a16, a17, a18, a19}; // A[2] for T0
  uint8_t a2_t1[4] = {a20, a21, a22, a23}; // A[2] for T1
  uint8_t a2_t2[4] = {a24, a25, a26, a27}; // A[2] for T2
  uint8_t a2_t3[4] = {a28, a29, a30, a31}; // A[2] for T3

  uint8_t b0_t0[4] = {b0, b1, b2, b3};     // B[0] for T0
  uint8_t b0_t1[4] = {b4, b5, b6, b7};     // B[0] for T1
  uint8_t b0_t2[4] = {b8, b9, b10, b11};   // B[0] for T2
  uint8_t b0_t3[4] = {b12, b13, b14, b15}; // B[0] for T3

  uint8_t b1_t0[4] = {b16, b17, b18, b19}; // B[1] for T0
  uint8_t b1_t1[4] = {b20, b21, b22, b23}; // B[1] for T1
  uint8_t b1_t2[4] = {b24, b25, b26, b27}; // B[1] for T2
  uint8_t b1_t3[4] = {b28, b29, b30, b31}; // B[1] for T3

  if (threadIdx.x == 0)
  {
    memcpy(&A[0], a0_t0, 4);
    memcpy(&A[2], a2_t0, 4);

    memcpy(&B[0], b0_t0, 4);
    memcpy(&B[1], b1_t0, 4);
  }

  if (threadIdx.x == 1)
  {
    memcpy(&A[0], a0_t1, 4);
    memcpy(&A[2], a2_t1, 4);

    memcpy(&B[0], b0_t1, 4);
    memcpy(&B[1], b1_t1, 4);
  }

  if (threadIdx.x == 2)
  {
    memcpy(&A[0], a0_t2, 4);
    memcpy(&A[2], a2_t2, 4);

    memcpy(&B[0], b0_t2, 4);
    memcpy(&B[1], b1_t2, 4);
  }

  if (threadIdx.x == 3)
  {
    memcpy(&A[0], a0_t3, 4);
    memcpy(&A[2], a2_t3, 4);

    memcpy(&B[0], b0_t3, 4);
    memcpy(&B[1], b1_t3, 4);
  }

#if USE_E4M3
  asm("mma.sync.aligned.m16n8k32.row.col.f32.e4m3.e4m3.f32 "
      "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
      : "=f"(D[0]), "=f"(D[1]), "=f"(D[2]), "=f"(D[3])
      : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]),
        "r"(B[0]), "r"(B[1]),
        "f"(C[0]), "f"(C[1]), "f"(C[2]), "f"(C[3]));
#endif

#if USE_E5M2
  asm("mma.sync.aligned.m16n8k32.row.col.f32.e5m2.e5m2.f32 "
      "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
      : "=f"(D[0]), "=f"(D[1]), "=f"(D[2]), "=f"(D[3])
      : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]),
        "r"(B[0]), "r"(B[1]),
        "f"(C[0]), "f"(C[1]), "f"(C[2]), "f"(C[3]));
#endif

  // Read result directly from D[0]
  fp32_ tc_result;
  tc_result.f = D[0];

  // check result using T0
#if DEBUG
  if (threadIdx.x == 0)
  {
    printf("Tensor Core result : %.20f, hex=%08X\n", tc_result.f, tc_result.i);
  }
#endif

  __syncthreads(); // ensure the 1st mma is done

  //------------------------------------------------------------------------//
  // second mma call :   add C
  //------------------------------------------------------------------------//

  // reset
  D[0] = 0.f;
  D[1] = 0.f;
  D[2] = 0.f;
  D[3] = 0.f;
  C[0] = 0.f;
  C[1] = 0.f;
  C[2] = 0.f;
  C[3] = 0.f;
  A[0] = 0;
  A[1] = 0;
  A[2] = 0;
  A[3] = 0;
  B[0] = 0;
  B[1] = 0;

  // Set Init C
  fp32_ c_fp32_hex;
  c_fp32_hex.i = test_init_c[0];
  C[0] = c_fp32_hex.f;

  // set a to 0
  a0 = 0;
  a1 = 0;
  a2 = 0;
  a3 = 0;
  a4 = 0;
  a5 = 0;
  a6 = 0;
  a7 = 0;
  a8 = 0;
  a9 = 0;
  a10 = 0;
  a11 = 0;
  a12 = 0;
  a13 = 0;
  a14 = 0;
  a15 = 0;

  a16 = 0;
  a17 = 0;
  a18 = 0;
  a19 = 0;
  a20 = 0;
  a21 = 0;
  a22 = 0;
  a23 = 0;
  a24 = 0;
  a25 = 0;
  a26 = 0;
  a27 = 0;
  a28 = 0;
  a29 = 0;
  a30 = 0;
  a31 = 0;

  // set b to 0
  b0 = 0;
  b1 = 0;
  b2 = 0;
  b3 = 0;
  b4 = 0;
  b5 = 0;
  b6 = 0;
  b7 = 0;
  b8 = 0;
  b9 = 0;
  b10 = 0;
  b11 = 0;
  b12 = 0;
  b13 = 0;
  b14 = 0;
  b15 = 0;

  b16 = 0;
  b17 = 0;
  b18 = 0;
  b19 = 0;
  b20 = 0;
  b21 = 0;
  b22 = 0;
  b23 = 0;
  b24 = 0;
  b25 = 0;
  b26 = 0;
  b27 = 0;
  b28 = 0;
  b29 = 0;
  b30 = 0;
  b31 = 0;

  //--- a0 x b0 ---//
  a0 = (a0 | test_ops[0]);
  b0 = (b0 | test_ops[1]);
  //--- a1 x b1 ---//
  a1 = (a1 | test_ops[2]);
  b1 = (b1 | test_ops[3]);
  //--- a2 x b2 ---//
  a2 = (a2 | test_ops[4]);
  b2 = (b2 | test_ops[5]);
  //--- a3 x b3 ---//
  a3 = (a3 | test_ops[6]);
  b3 = (b3 | test_ops[7]);

  //--- a4 x b4 ---//
  a4 = (a4 | test_ops[8]);
  b4 = (b4 | test_ops[9]);
  //--- a5 x b5 ---//
  a5 = (a5 | test_ops[10]);
  b5 = (b5 | test_ops[11]);
  //--- a6 x b6 ---//
  a6 = (a6 | test_ops[12]);
  b6 = (b6 | test_ops[13]);
  //--- a7 x b7 ---//
  a7 = (a7 | test_ops[14]);
  b7 = (b7 | test_ops[15]);

  //--- a8 x b8 ---//
  a8 = (a8 | test_ops[16]);
  b8 = (b8 | test_ops[17]);
  //--- a9 x b9 ---//
  a9 = (a9 | test_ops[18]);
  b9 = (b9 | test_ops[19]);
  //--- a10 x b10 ---//
  a10 = (a10 | test_ops[20]);
  b10 = (b10 | test_ops[21]);
  //--- a11 x b11 ---//
  a11 = (a11 | test_ops[22]);
  b11 = (b11 | test_ops[23]);

  //--- a12 x b12 ---//
  a12 = (a12 | test_ops[24]);
  b12 = (b12 | test_ops[25]);
  //--- a13 x b13 ---//
  a13 = (a13 | test_ops[26]);
  b13 = (b13 | test_ops[27]);
  //--- a14 x b14 ---//
  a14 = (a14 | test_ops[28]);
  b14 = (b14 | test_ops[29]);
  //--- a15 x b15 ---//
  a15 = (a15 | test_ops[30]);
  b15 = (b15 | test_ops[31]);

  //--- a16 x b16 ---//
  a16 = (a16 | test_ops[32]);
  b16 = (b16 | test_ops[33]);
  //--- a17 x b17 ---//
  a17 = (a17 | test_ops[34]);
  b17 = (b17 | test_ops[35]);
  //--- a18 x b18 ---//
  a18 = (a18 | test_ops[36]);
  b18 = (b18 | test_ops[37]);
  //--- a19 x b19 ---//
  a19 = (a19 | test_ops[38]);
  b19 = (b19 | test_ops[39]);

  //--- a20 x b20 ---//
  a20 = (a20 | test_ops[40]);
  b20 = (b20 | test_ops[41]);
  //--- a21 x b21 ---//
  a21 = (a21 | test_ops[42]);
  b21 = (b21 | test_ops[43]);
  //--- a22 x b22 ---//
  a22 = (a22 | test_ops[44]);
  b22 = (b22 | test_ops[45]);
  //--- a23 x b23 ---//
  a23 = (a23 | test_ops[46]);
  b23 = (b23 | test_ops[47]);

  //--- a24 x b24 ---//
  a24 = (a24 | test_ops[48]);
  b24 = (b24 | test_ops[49]);
  //--- a25 x b25 ---//
  a25 = (a25 | test_ops[50]);
  b25 = (b25 | test_ops[51]);
  //--- a26 x b26 ---//
  a26 = (a26 | test_ops[52]);
  b26 = (b26 | test_ops[53]);
  //--- a27 x b27 ---//
  a27 = (a27 | test_ops[54]);
  b27 = (b27 | test_ops[55]);

  //--- a28 x b28 ---//
  a28 = (a28 | test_ops[56]);
  b28 = (b28 | test_ops[57]);
  //--- a29 x b29 ---//
  a29 = (a29 | test_ops[58]);
  b29 = (b29 | test_ops[59]);
  //--- a30 x b30 ---//
  a30 = (a30 | test_ops[60]);
  b30 = (b30 | test_ops[61]);
  //--- a31 x b31 ---//
  a31 = (a31 | test_ops[62]);
  b31 = (b31 | test_ops[63]);

  // Pass the input ops to tc inputs
  uint8_t a0_t0_[4] = {a0, a1, a2, a3};     // A[0] for T0
  uint8_t a0_t1_[4] = {a4, a5, a6, a7};     // A[0] for T1
  uint8_t a0_t2_[4] = {a8, a9, a10, a11};   // A[0] for T2
  uint8_t a0_t3_[4] = {a12, a13, a14, a15}; // A[0] for T3

  uint8_t a2_t0_[4] = {a16, a17, a18, a19}; // A[2] for T0
  uint8_t a2_t1_[4] = {a20, a21, a22, a23}; // A[2] for T1
  uint8_t a2_t2_[4] = {a24, a25, a26, a27}; // A[2] for T2
  uint8_t a2_t3_[4] = {a28, a29, a30, a31}; // A[2] for T3

  uint8_t b0_t0_[4] = {b0, b1, b2, b3};     // B[0] for T0
  uint8_t b0_t1_[4] = {b4, b5, b6, b7};     // B[0] for T1
  uint8_t b0_t2_[4] = {b8, b9, b10, b11};   // B[0] for T2
  uint8_t b0_t3_[4] = {b12, b13, b14, b15}; // B[0] for T3

  uint8_t b1_t0_[4] = {b16, b17, b18, b19}; // B[1] for T0
  uint8_t b1_t1_[4] = {b20, b21, b22, b23}; // B[1] for T1
  uint8_t b1_t2_[4] = {b24, b25, b26, b27}; // B[1] for T2
  uint8_t b1_t3_[4] = {b28, b29, b30, b31}; // B[1] for T3

  if (threadIdx.x == 0)
  {
    memcpy(&A[0], a0_t0_, 4);
    memcpy(&A[2], a2_t0_, 4);

    memcpy(&B[0], b0_t0_, 4);
    memcpy(&B[1], b1_t0_, 4);
  }

  if (threadIdx.x == 1)
  {
    memcpy(&A[0], a0_t1_, 4);
    memcpy(&A[2], a2_t1_, 4);

    memcpy(&B[0], b0_t1_, 4);
    memcpy(&B[1], b1_t1_, 4);
  }

  if (threadIdx.x == 2)
  {
    memcpy(&A[0], a0_t2_, 4);
    memcpy(&A[2], a2_t2_, 4);

    memcpy(&B[0], b0_t2_, 4);
    memcpy(&B[1], b1_t2_, 4);
  }

  if (threadIdx.x == 3)
  {
    memcpy(&A[0], a0_t3_, 4);
    memcpy(&A[2], a2_t3_, 4);

    memcpy(&B[0], b0_t3_, 4);
    memcpy(&B[1], b1_t3_, 4);
  }

#if USE_E4M3
  asm("mma.sync.aligned.m16n8k32.row.col.f32.e4m3.e4m3.f32 "
      "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
      : "=f"(D[0]), "=f"(D[1]), "=f"(D[2]), "=f"(D[3])
      : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]),
        "r"(B[0]), "r"(B[1]),
        "f"(C[0]), "f"(C[1]), "f"(C[2]), "f"(C[3]));
#endif

#if USE_E5M2
  asm("mma.sync.aligned.m16n8k32.row.col.f32.e5m2.e5m2.f32 "
      "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};\n"
      : "=f"(D[0]), "=f"(D[1]), "=f"(D[2]), "=f"(D[3])
      : "r"(A[0]), "r"(A[1]), "r"(A[2]), "r"(A[3]),
        "r"(B[0]), "r"(B[1]),
        "f"(C[0]), "f"(C[1]), "f"(C[2]), "f"(C[3]));
#endif

  // Read result directly from D[0]
  fp32_ tc_result1;
  tc_result1.f = D[0];

  // check result using T0
#if DEBUG
  if (threadIdx.x == 0)
  {
    printf("Tensor Core result : %.20f, hex=%08X\n", tc_result1.f, tc_result1.i);
  }
#endif

  //------------------------------------------------------------------------//
  // save tc results
  //------------------------------------------------------------------------//
  if (threadIdx.x == 0)
  {
    result_gpu[0] = tc_result.i;
    result_gpu[1] = tc_result1.i;
  }

  //---------------------//
  // avoid optimze code away
  //---------------------//
  C[0] += D[0];
  C[1] += D[1];
  C[1] += C[0];
  C[0] += C[1];

  // copy c0,c1
  memcpy(&buf_fp32[threadIdx.x * 4], C, 16); // 4 of 32 bits = 16 bytes = 4 fp32 values
}
*/

// __global__ void warmupKernel() {
//   __shared__ int s[100];
//   s[0] += s[1];
// }



// input file 
// 65 input per row:   c +  32 of a/b
int main(int argc, char **argv)
{
  //------------------------------------------------------------------------//
  // Read commandlines
  //------------------------------------------------------------------------//
  if (argc != 2)
  {
    std::cerr << "\nUsage: " << argv[0] << " <filename>" << std::endl;
    return 1;
  }

  //------------------------------------------------------------------------//
  // Read all test cases
  //------------------------------------------------------------------------//
  std::vector<uint32_t> allTests_c;              // input c
  std::vector<std::vector<uint8_t>> allTests_ab; // input a/b

  std::cout << "file : " << argv[1] << std::endl;

  std::ifstream file(argv[1], std::ifstream::ate | std::ifstream::binary);
  if (!file.is_open())
  {
    std::cerr << "Unable to open file " << argv[1] << std::endl;
    return 1;
    }

    // Get the size of the file
    std::streamsize fileSize = file.tellg();
    file.seekg(0, std::ios::beg);

    std::cout << "Read all the tests cases ... " << std::endl;
    std::string line;
    while (getline(file, line))
    {
        // read line :    c a0 b0 a1 b1 ....  aN bN
        std::istringstream iss(line);
        std::vector<uint8_t> numbers_ab;  // current line for a/b
        std::string hexStr;

        // read c first
        iss >> hexStr;
        uint32_t num_c = static_cast<uint32_t>(std::stoul(hexStr, nullptr, 16));
        allTests_c.push_back(num_c);  // store current line for C

        // read a/b
        while (iss >> hexStr)
        {
            uint8_t num = static_cast<uint8_t>(std::stoul(hexStr, nullptr, 16));
            numbers_ab.push_back(num);
        }

        allTests_ab.push_back(numbers_ab); // store current line for a and b 
    }

    std::cout << std::endl;

    file.close();

    //------------------------------------------------------------------------//
    // Check first line : c + 32x{a(i), b(i)}
    //------------------------------------------------------------------------//
#if DEBUG
    printf("\nCheck first line of input file:\n");
    printf("%08X ", allTests_c[0]);
    for (int i = 0; i < 64; i++)
    {
        printf("%02X ", allTests_ab[0][i]);
    }
    printf("\n\n");
#endif

    //------------------------------------------------------------------------//
    // Run all test cases
    //------------------------------------------------------------------------//
    std::cout << "\nRun Tensor Core Tests" << std::endl;

    // prepare results
    int totalNum = static_cast<int>(allTests_ab.size());
    std::vector<std::vector<uint32_t>> allTests_results(totalNum);

    for (int i = 0; i < totalNum; i++)
    {
      // each test inputs
      uint32_t current_test_c = allTests_c[i];
      std::vector<uint8_t> current_test_ab = allTests_ab[i];

      // output
      std::vector<uint32_t> current_result;

      //--------------------------------------------------------------------//
      // run tensor core test
      //--------------------------------------------------------------------//
      runTest<K32>(current_test_ab, current_test_c, current_result);

#if DEBUG
      // printf("%08X %08X\n", current_result[0], current_result[1]);
#endif
      allTests_results[i] = current_result;

    }

/*
      for (int kernel_num : {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11}) {
        // for (int kernel_num : {0, 11}) {
        // Give the GPU some rest to avoid thermal throttling
        sleep(5);
        std::cout << "KERNEL " << kernel_num << std::endl;
        // Verify against cuBLAS. Also serves as a warmup step.
        if (run_verif) {
          memset(C, 0, sizeof(bf16) * max_size * max_size);
          cudaCheck(hipMemcpy(dC, C, sizeof(bf16) * max_size * max_size, hipMemcpyHostToDevice));
          cudaCheck(hipMemcpy(dC_ref, C, sizeof(bf16) * max_size * max_size, hipMemcpyHostToDevice));
          memset(DB, ~0, sizeof(int) * max_size * 128);
          cudaCheck(hipMemcpy(dDB, DB, sizeof(int) * max_size * 128,
            hipMemcpyHostToDevice));
          run_kernel(0, m, n, k, dA, dB, dC_ref); // cuBLAS
          run_kernel(kernel_num, m, n, k, dA, dB, dC, dDB); // Executes the kernel, modifies the result matrix
          cudaCheck(hipDeviceSynchronize());
          cudaCheck(hipGetLastError()); // Check for async errors during kernel run
          hipMemcpy(C, dC, sizeof(bf16) * max_size * max_size, hipMemcpyDeviceToHost);
          hipMemcpy(C_ref, dC_ref, sizeof(bf16) * max_size * max_size, hipMemcpyDeviceToHost);

          if (kernel_num > 1 && !verify_matrix(C_ref, C, m * n)) {
            std::cout << "~~~~~~~~~~~~~~~~ Failed to pass the correctness verification against cuBLAS. ~~~~~~~~~~~~~~~~" << std::endl;
            printf("%f\n", __bfloat162float(C_ref[m]));
          }

          hipMemcpy(DB, dDB, sizeof(int) * max_size * 8, hipMemcpyDeviceToHost);

      */
    return 0;
};


//----------------------------------------------------------------------------//
// host code to prepare test
//----------------------------------------------------------------------------//
template <int TILE_K>
void runTest(std::vector<uint8_t> current_test_ab,
			 uint32_t current_test_c,
			 std::vector<uint32_t> &current_result)
{
	//------------------------------------------------------------------------//
	// total workload size M 64 x N 8 x K 32
	//------------------------------------------------------------------------//
	long max_size = 64;
	long M = 64, N = 8, K = 32;

	//------------------------------------------------------------------------//
	// prepare host buffers
	//------------------------------------------------------------------------//
	uint8_t *hA = nullptr;
	uint8_t *hB = nullptr;

	uint32_t *hC = nullptr;
	uint32_t *hD = nullptr;

	size_t sizeA = M * K;
	size_t sizeB = N * K;
	size_t sizeCD = M * N;

	hA = (uint8_t *)malloc(sizeof(uint8_t) * sizeA);
	hB = (uint8_t *)malloc(sizeof(uint8_t) * sizeB);

	hC = (uint32_t *)malloc(sizeof(uint32_t) * sizeCD);
	hD = (uint32_t *)malloc(sizeof(uint32_t) * sizeCD);

	// init to 0
	memset(hA, 0, sizeof(uint8_t) * sizeA);
	memset(hB, 0, sizeof(uint8_t) * sizeB);

	memset(hC, 0, sizeof(uint32_t) * sizeCD);
	memset(hD, 0, sizeof(uint32_t) * sizeCD);

	//------------------------------------------------------------------------//
	// read/set up data on cpu
	//------------------------------------------------------------------------//
	std::cout << "Read inputs a/b " << std::endl;
	for (int i = 0; i < TILE_K; i++)
	{
		hA[i] = current_test_ab[i * 2];		//  read a
		hB[i] = current_test_ab[i * 2 + 1]; //  read b
	}

	std::cout << "Read input C" << std::endl;
	hC[0] = current_test_c;

	// // cpu buffer: two results in fp32
	// // 1st : dotproduct of (a, b)
	// // 2nd : dotproduct of (a, b) + C
	// uint32_t *result_cpu = 0;
	// result_cpu = (uint32_t *)malloc(sizeof(uint32_t) * 2);

	//------------------------------------------------------------------------//
	// gpu buffer
	//------------------------------------------------------------------------//
	uint8_t *dA = nullptr;
	uint8_t *dB = nullptr;
	uint32_t *dC = nullptr;
	uint32_t *dD = nullptr;

	hipMalloc((void **)&dA, sizeof(uint8_t) * sizeA);
	hipMalloc((void **)&dB, sizeof(uint8_t) * sizeB);

	hipMalloc((void **)&dC, sizeof(uint32_t) * sizeCD);
	hipMalloc((void **)&dD, sizeof(uint32_t) * sizeCD);

	// float *buf_fp32 = 0;
	// half *buf_fp16 = 0;
	// hipMalloc((void **)&buf_fp32, sizeof(float) * 1024);
	// hipMalloc((void **)&buf_fp16, sizeof(half) * 1024);


	// // output buffer
	// hipMalloc((void **)&result_gpu, sizeof(uint32_t) * 2);

	// h2d : copy input ops to gpu
	hipMemcpy(dA, hA, sizeof(uint8_t) * sizeA, hipMemcpyHostToDevice);
	hipMemcpy(dB, hB, sizeof(uint8_t) * sizeB, hipMemcpyHostToDevice);

	hipMemcpy(dC, hC, sizeof(uint32_t) * sizeCD, hipMemcpyHostToDevice);
	hipMemcpy(dD, hD, sizeof(uint32_t) * sizeCD, hipMemcpyHostToDevice);

	constexpr int BM = 64;
	constexpr int BN = 8;
	constexpr int BK = 32;
	constexpr int NUM_THREADS = 128;

	CUtensorMap *d_tma_map_A = 0;
	CUtensorMap *d_tma_map_B = 0;

	//   d_tma_map_A = allocate_and_create_tensor_map<BM, BK>(dA, m / BM, k / BK);
	//   d_tma_map_B = allocate_and_create_tensor_map<BN, BK>(dB, m / BN, k / BK);

	//------------------------------------------------------------------------//
	// 1 warpgroup = 128 threads
	//------------------------------------------------------------------------//
	//   kernel_wgmma_FP8<<<1, 128>>>(buf_fp32, buf_fp16, test_inputs_ab, test_inputs_c, result_gpu);

	/*
	  // d2h : copy results back to host
	  hipMemcpy(result_cpu, result_gpu, sizeof(uint32_t) * 2, hipMemcpyDeviceToHost);

	  // check value
	#if DEBUG
	  printf("%08X %08X\n", result_cpu[0], result_cpu[1]);
	#endif

	  current_result.push_back(result_cpu[0]);
	  current_result.push_back(result_cpu[1]);

	  if (buf_fp32)
	  {
		hipFree(buf_fp32);
	  }

	  if (buf_fp16)
	  {
		hipFree(buf_fp16);
	  }

	  if (test_inputs_ab)
	  {
		hipFree(test_inputs_ab);
	  }

	  if (test_inputs_c)
	  {
		hipFree(&test_inputs_c);
	  }

	  if (result_gpu)
	  {
		hipFree(result_gpu);
	  }
	  */

	//   free(result_cpu);

	hipFree(dA);
	hipFree(dB);
	hipFree(dC);
	hipFree(dD);

	free(hA);
	free(hB);
	free(hC);
	free(hD);

	hipDeviceSynchronize();
}